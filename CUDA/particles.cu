#include "hip/hip_runtime.h"
#include "particles.hpp"

#include <iostream>
#include <algorithm>
#include <immintrin.h>
#include <hip/hip_runtime.h>

Particles::Particles() {
    this->timeStep = 0.0f;
}

/**
 * @brief Construct a new Particles:: Particles object
 * 
 * @param config Config structure
 */
Particles::Particles(SimConfig_t *config) {
    this->timeStep = config->timeStep;
    this->width = config->simWidth;
    this->height = config->simHeight;

    this->numParticles = config->numParticles;
    float particleSize = config->particleSize;
    float particleMass = config->particleMass;
    float maxSpeed = config->maxSpeed;

    temp_particles = new float[this->numParticles * 4]();

    for (uint i = 0; i < this->numParticles; i++) {
        bool intersecting = true;
        while (intersecting) {
            float x = particleSize + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(width - 2 * particleSize)));
            float y = particleSize + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(height - 2 * particleSize)));
            float dx = -maxSpeed + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(2 * maxSpeed)));
            float dy = -maxSpeed + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(2 * maxSpeed)));
            Particle *newParticle = new Particle(x, y, dx, dy, particleSize, particleMass);
            newParticle->particleNum = i;

            if (particles.empty()) {
                intersecting = false;
                particles.push_back(*newParticle);

                temp_particles[i * 4] = newParticle->get_x();
                temp_particles[i * 4 + 1] = newParticle->get_y();
                temp_particles[i * 4 + 2] = newParticle->get_dx();
                temp_particles[i * 4 + 3] = newParticle->get_dy();
            } else {
                bool isIntersecting = false;
                vector<Particle>::iterator it;
                for (it = particles.begin(); it != particles.end(); it++) {
                    if (checkCollision(newParticle, &(*it))) {
                        isIntersecting = true;
                    }
                }
                if (!isIntersecting) {
                    intersecting = false;
                    particles.push_back(*newParticle);

                    temp_particles[i * 4] = newParticle->get_x();
                    temp_particles[i * 4 + 1] = newParticle->get_y();
                    temp_particles[i * 4 + 2] = newParticle->get_dx();
                    temp_particles[i * 4 + 3] = newParticle->get_dy();
                } else {
                    delete newParticle;
                }
            }
        }
    }

    // Allocate device particle storage
    hipMalloc((void **)&d_particles, sizeof(float) * this->numParticles * 4); 
    hipMemcpy(d_particles, temp_particles, sizeof(float) * this->numParticles * 4, hipMemcpyHostToDevice);

    this->gridSize = config->particleSize * 2;

    this->numRows = this->height / this->gridSize;
    this->numCols = this->width / this->gridSize;

    // std::cout << this->numRows << std::endl;

    // Allocate grid
    hipMalloc((void **)&gridCounters, sizeof(uint) * this->numRows * this->numCols);
    hipMalloc((void **)&gridCells, sizeof(uint) * this->numRows * this->numCols * MAX_PARTICLES_PER_CELL);

    // Allocate blank grid host side
    blankCounters = new uint[this->numRows * this->numCols]();
    blankCells = new uint[this->numRows * this->numCols * MAX_PARTICLES_PER_CELL]();
}

/**
 * @brief Increments the simulation by updating the current time.
 * 
 */
void Particles::updateTime() {
    this->currentTime += this->timeStep;
}

__global__ void d_updateGrid(float *particles, uint *counters, uint *cells, float size, uint cols) {
    uint particle = blockIdx.x * blockDim.x + threadIdx.x;

    // Figure out which grid we're in
    uint gridX = floor(particles[particle * 4] / size);
    uint gridY = floor(particles[particle * 4 + 1] / size);

    // Add to counter and cell
    atomicInc(&counters[gridY * cols + gridX]);
    for (uint i = 0; i < 4; i++) {
        __threadfence();
        if (cells[gridY * cols + gridX] == 0) {
            cells[gridY * cols + gridX] = particle;
            break;
        }
    }
}

void Particles::updateGrid() {
    hipMemcpy(gridCounters, blankCounters, sizeof(uint) * this->numRows * this->numCols, hipMemcpyHostToDevice);
    hipMemcpy(gridCells, blankCells, sizeof(uint) * this->numRows * this->numCols * MAX_PARTICLES_PER_CELL, hipMemcpyHostToDevice);

    d_updateGrid<<<(this->numParticles + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(d_particles, gridCounters, gridCells, this->particleSize, this->numCols);
}

__global__ d_updateCollisions(float *particles, uint *counters, uint *cells, float size) {
    uint particle = blockIdx.x * blockDim.x + threadIdx.x;

    // Check current cell
    uint gridX = floor(particles[particle * 4] / size);
    uint gridY = floor(particles[particle * 4 + 1] / size);

    int addX = particles[particle * 4] >= size * (gridX + 0.5) ? 1 : -1;
    int addY = particles[particle * 4 + 1] >= size * (gridY + 0.5) ? 1 : -1;

    __threadfence();
    if (counters[gridY * cols + gridX] != 0) {
        // do stuff
    } else if (counters[gridY * cols + gridX + addX] != 0) {
        // do more stuff
    } else if (counters[(gridY + addY) * cols + gridX] != 0) {
        // do even more stuff
    } else if (counters[(gridY + addY) * cols + gridX + addX] != 0) {
        // do even even more stuff
    }

    
}

void Particles::updateCollisions() {


    // uint i, j;
    // for (i = 0; i < numRows; i++) {
    //     auto row = rows.at(i);
    //     for (j = 0; j < numRows; j++) {
    //         auto cell = row.cells.at(j);
    //         if (cell.elements.size() > 1) {
    //             for (int i1 : cell.elements) {
    //                 auto p1 = &(particles.at(i1));
    //                 for (int i2 : cell.elements) {
    //                     auto p2 = &(particles.at(i2));
    //                     if (i1 != i2 && !p1->hasCollided && !p2->hasCollided) {
    //                         if (checkCollision(p1, p2)) {
    //                             // https://gamedevelopment.tutsplus.com/tutorials/when-worlds-collide-simulating-circle-circle-collisions--gamedev-769
    //                             float newXVel1 = (p1->get_dx() * (p1->get_mass() - p2->get_mass()) + (2 * p2->get_mass() * p2->get_dx())) / (p1->get_mass() + p2->get_mass());
    //                             float newXVel2 = (p2->get_dx() * (p2->get_mass() - p1->get_mass()) + (2 * p1->get_mass() * p1->get_dx())) / (p1->get_mass() + p2->get_mass());
    //                             float newYVel1 = (p1->get_dy() * (p1->get_mass() - p2->get_mass()) + (2 * p2->get_mass() * p2->get_dy())) / (p1->get_mass() + p2->get_mass());
    //                             float newYVel2 = (p2->get_dy() * (p2->get_mass() - p1->get_mass()) + (2 * p1->get_mass() * p1->get_dy())) / (p1->get_mass() + p2->get_mass());

    //                             p1->set_dx(newXVel1);
    //                             p2->set_dx(newXVel2);
    //                             p1->set_dy(newYVel1);
    //                             p2->set_dy(newYVel2);

    //                             p1->hasCollided = true;
    //                             p2->hasCollided = true;
    //                         }
    //                     }
    //                 }
    //             }
    //         }
    //     }
    // }
}

void Particles::updateMovements() {
    vector<Particle>::iterator it;
    for (it = particles.begin(); it != particles.end(); it++) {
        float x = it->get_x();
        float y = it->get_y();
        float radius = it->get_radius();

        if (x < radius || x > width - radius) {
            it->set_dx(-1 * it->get_dx());
        }
        if (y < radius || y > width - radius) {
            it->set_dy(-1 * it->get_dy());
        }

        it->update(timeStep);
        it->hasCollided = false;
    }
}

inline bool Particles::checkAABBCircle(float p1x, float p1y, float p1r, float p2x, float p2y, float p2r) {
    return p1x + p1r + p2r > p2x
        && p1x < p2x + p1r + p2r
        && p1y + p1r + p2r > p2y
        && p1y < p2y + p1r + p2r;
}

inline bool Particles::checkAABBRect(float p1x, float p1y, float p1w, float p1h, float p2x, float p2y, float p2w, float p2h) {
    return p1x + p1w + p2w > p2x
        && p1x < p2x + p1w + p2w
        && p1y + p1h + p2h > p2y
        && p1y < p2y + p1h + p2h;
}

/**
 * @brief Checks for collisions between two particles
 * 
 * @param p1 Pointer to first particle
 * @param p2 Pointer to second particle
 * @return true 
 * @return false 
 */
bool Particles::checkCollision(Particle *p1, Particle *p2) {
    // AABB collision check
    if (checkAABBCircle(p1->get_x(), p1->get_y(), p1->get_radius(), p2->get_x(), p2->get_y(), p2->get_radius())) {
        // Actual collision check
        float distance = sqrtf(
            ((p1->get_x() - p2->get_x()) * (p1->get_x() - p2->get_x()))
            + ((p1->get_y() - p2->get_y()) * (p1->get_y() - p2->get_y()))
        );
        if (distance < p1->get_radius() + p2->get_radius()) {
            return true;
        }
    }
    return false;
}